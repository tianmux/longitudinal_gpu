#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
    This example demonstrates how to use the Cuda OpenGL bindings to
    dynamically modify a vertex buffer using a Cuda kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with Cuda
    3. Map the VBO for writing from Cuda
    4. Run Cuda kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include "helper_gl.h"
#if defined (__APPLE__) || defined(MACOSX)
  #pragma clang diagnostic ignored "-Wdeprecated-declarations"
  #include <GLUT/glut.h>
  #ifndef glutCloseFunc
  #define glutCloseFunc glutWMCloseFunc
  #endif
#else
#include <GL/freeglut.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD          0.30f
#define REFRESH_DELAY     5 //ms

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width  = 1024;
const unsigned int window_height = 1024;


const unsigned int Nring = 1<<6;
const unsigned int n_per_ring = 1<<10;
const unsigned int Npar = n_per_ring*Nring;
const unsigned int n_per_show = 10;
double Qx = 3.666;
double nux = 2.0*3.1415926*Qx;
double sinnux = sin(nux);
double cosnux = cos(nux);
double Qy = 3.7;
double nuy = 2.0*3.1415926*Qy;
double sinnuy = sin(nuy);
double cosnuy = cos(nuy);
double S = 0.0;
double bt = 1.0;
double v1 = 0;//used to manually control the voltage.
double v2 = 1;

unsigned int initial = 0;

// vbo variables
GLuint vbo;
GLuint vbox;
struct hipGraphicsResource *cuda_vbo_resource;
struct hipGraphicsResource *cuda_vbox_resource;
void *d_vbo_buffer = NULL;
void *d_vbox_buffer = NULL;
float g_fAnim = 0.0;

// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -1;

StopWatchInterface *timer = NULL;

// Auto-Verification Code
int fpsCount = 0;        // FPS count for averaging
int fpsLimit = 1;        // FPS limit for sampling
float GFLOPS = 0;        // GFLOPS
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv, char *ref_file);
void cleanup();

// GL functionality
bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags);
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res);

// rendering callbacks
void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

// Cuda functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGL (VBO)";

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////

__global__ void init_kernel(double4 *pos, unsigned int Npar)
{
    unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;

    double xx = 0.5*(double)(blockIdx.x+0.0)/(double)Nring*cosf(2*3.1415926*(double)threadIdx.x/(double)blockDim.x);
    double yy = 0.5*(double)(blockIdx.x+0.0)/(double)Nring*cosf(2*3.1415926*(double)threadIdx.x/(double)blockDim.x);
    double px = 0.5*(double)(blockIdx.x+0.0)/(double)Nring*sinf(2*3.1415926*(double)threadIdx.x/(double)blockDim.x);
    double py = 0.5*(double)(blockIdx.x+0.0)/(double)Nring*sinf(2*3.1415926*(double)threadIdx.x/(double)blockDim.x);

    // write output vertex
    pos[n] = make_double4(xx,px,yy,py);
}

__global__ void simple_vbo_kernel(double4 *pos, double v1, double v2)
{
    unsigned int n = blockIdx.x*blockDim.x + threadIdx.x;
    double x = pos[n].x;
    double px = pos[n].y;

    for(unsigned int i = 0;i<n_per_show;++i){
        
        px = px+1e-3*(1-v1)*(sinf(x))+1e-3*(1-v2)*(sinf(2.0*x)-1);
        x = x-0.001*px;
    }
/*
    x = cosnux*xx+bt*sinnux*pxx;
    y = cosnuy*yy+bt*sinnuy*pyy;
    px = -1.0/bt*sinnux*xx+cosnux*pxx-0.5*S*(x*x-y*y);
    py = -1.0/bt*sinnuy*yy+cosnuy*pyy-S*x*y;
*/
    //if (x > 5 || y >5 || px>5||py>5){
    //    x = 0;
    //    y = 0;
    //    px = 0;
    //   py = 0;
    //}
    // write output vertex
    pos[n] = make_double4(x,px,0,0);
}


void launch_kernel(double4 *pos,  unsigned int Npar,double nu,double S,double bt)
{
    // execute the kernel
    dim3 block(1024, 1, 1);
    dim3 grid(Npar/ block.x>1?Npar/ block.x:1, 1, 1);
    simple_vbo_kernel<<< grid, block>>>(pos,v1,v2);
}

bool checkHW(char *name, const char *gpuType, int dev)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    strcpy(name, deviceProp.name);

    if (!STRNCASECMP(deviceProp.name, gpuType, strlen(gpuType)))
    {
        return true;
    }
    else
    {
        return false;
    }
}

int findGraphicsGPU(char *name)
{
    int nGraphicsGPU = 0;
    int deviceCount = 0;
    bool bFoundGraphics = false;
    char firstGraphicsName[256], temp[256];

    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess)
    {
        printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        printf("> FAILED %s sample finished, exiting...\n", sSDKsample);
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0)
    {
        printf("> There are no device(s) supporting CUDA\n");
        return false;
    }
    else
    {
        printf("> Found %d CUDA Capable Device(s)\n", deviceCount);
    }

    for (int dev = 0; dev < deviceCount; ++dev)
    {
        bool bGraphics = !checkHW(temp, (const char *)"Tesla", dev);
        printf("> %s\t\tGPU %d: %s\n", (bGraphics ? "Graphics" : "Compute"), dev, temp);

        if (bGraphics)
        {
            if (!bFoundGraphics)
            {
                strcpy(firstGraphicsName, temp);
            }

            nGraphicsGPU++;
        }
    }

    if (nGraphicsGPU)
    {
        strcpy(name, firstGraphicsName);
    }
    else
    {
        strcpy(name, "this hardware");
    }

    return nGraphicsGPU;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

#if defined(__linux__)
    setenv ("DISPLAY", ":0", 0);
#endif

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            // In this mode, we are running non-OpenGL and doing a compare of the VBO was generated correctly
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    printf("\n");

    runTest(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (g_TotalErrors == 0) ? "OK" : "ERROR!");
    exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);
        GFLOPS = 18.0f*Npar*n_per_show/(sdkGetAverageTimerValue(&timer) / 1000.f)/1000000000;
        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "#Particles:%4d; #Turns: %4d; GFLOPS: %4.4f, Time per turn (us): %4.4f", Npar,frameCount*n_per_show,GFLOPS,sdkGetAverageTimerValue(&timer)/n_per_show*1000);
    glutSetWindowTitle(fps);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize GL
////////////////////////////////////////////////////////////////////////////////
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("Cuda GL Interop (VBO)");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    // initialize necessary OpenGL extensions
    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    // default initialization
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    // viewport
    glViewport(0, 0, window_width, window_height);

    // projection
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)window_width / (GLfloat) window_height, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}


////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file)
{
    // Create the CUTIL timer
    sdkCreateTimer(&timer);

    // command line mode only
    if (ref_file != NULL)
    {
        // This will pick the best possible CUDA capable device
        int devID = findCudaDevice(argc, (const char **)argv);

        // create VBO
        checkCudaErrors(hipMalloc((void **)&d_vbo_buffer, Npar*4*sizeof(double)));

        // run the cuda part
        runAutoTest(devID, argv, ref_file);

        // check result of Cuda step
        checkResultCuda(argc, argv, vbo);

        hipFree(d_vbo_buffer);
        d_vbo_buffer = NULL;
    }
    else
    {
        // First initialize OpenGL context, so we can properly set the GL for CUDA.
        // This is necessary in order to achieve optimal performance with OpenGL/CUDA interop.
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        if (checkCmdLineFlag(argc, (const char **)argv, "device"))
        {
            if (gpuGLDeviceInit(argc, (const char **)argv) == -1)
            {
                return false;
            }
        }
        else
        {
            cudaGLSetGLDevice(gpuGetMaxGflopsDeviceId());
        }

        // register callbacks
        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
#if defined (__APPLE__) || defined(MACOSX)
        atexit(cleanup);
#else
        glutCloseFunc(cleanup);
#endif

        // create VBO
        createVBO(&vbo, &cuda_vbo_resource, cudaGraphicsMapFlagsWriteDiscard);

        // run the cuda part
        runCuda(&cuda_vbo_resource);

        // start rendering mainloop
        glutMainLoop();
    }

    return true;
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource)
{
    // map OpenGL buffer object for writing from CUDA
    double4 *dptr;
    checkCudaErrors(hipGraphicsMapResources(1, vbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                                         *vbo_resource));
    //printf("CUDA mapped VBO: May access %ld bytes\n", num_bytes);

    // execute the kernel
    //    dim3 block(8, 8, 1);
    //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);
    if (initial == 0){
        init_kernel<<<Npar/n_per_ring,n_per_ring>>>(dptr,Npar);
        initial =1;
    }
    launch_kernel(dptr, Npar, nux, S, bt);

    // unmap buffer object
    checkCudaErrors(hipGraphicsUnmapResources(1, vbo_resource, 0));
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file)
{
    char *reference_file = NULL;
    void *imageData = malloc(Npar*sizeof(double));

    // execute the kernel
    launch_kernel((double4 *)d_vbo_buffer, Npar, nux,S,bt);

    hipDeviceSynchronize();
    getLastCudaError("launch_kernel failed");

    checkCudaErrors(hipMemcpy(imageData, d_vbo_buffer, Npar*sizeof(double), hipMemcpyDeviceToHost));

    sdkDumpBin2(imageData, Npar*sizeof(double), "simpleGL.bin");
    reference_file = sdkFindFilePath(ref_file, argv[0]);

    if (reference_file &&
        !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                                Npar*sizeof(double),
                                MAX_EPSILON_ERROR, THRESHOLD, pArgv[0]))
    {
        g_TotalErrors++;
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Create VBO
////////////////////////////////////////////////////////////////////////////////
void createVBO(GLuint *vbo, struct hipGraphicsResource **vbo_res,
               unsigned int vbo_res_flags)
{
    assert(vbo);

    // create buffer object
    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    // initialize buffer object
    unsigned int size = Npar * 4 * sizeof(double);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);

    glBindBuffer(GL_ARRAY_BUFFER, 0);

    // register this buffer object with CUDA
    checkCudaErrors(hipGraphicsGLRegisterBuffer(vbo_res, *vbo, vbo_res_flags));

    SDK_CHECK_ERROR_GL();
}

////////////////////////////////////////////////////////////////////////////////
//! Delete VBO
////////////////////////////////////////////////////////////////////////////////
void deleteVBO(GLuint *vbo, struct hipGraphicsResource *vbo_res)
{

    // unregister this buffer object with CUDA
    checkCudaErrors(hipGraphicsUnregisterResource(vbo_res));

    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    bt-=0.0001;
    runCuda(&cuda_vbo_resource);
    g_fAnim += 0.01f;
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);


    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translate_z);
    glRotatef(rotate_x, 1.0, 0.0, 0.0);
    glRotatef(rotate_y, 0.0, 1.0, 0.0);
    glPointSize(4.0);
    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(2, GL_DOUBLE, 4*sizeof(double), 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_POINTS, 0, Npar);
    glDisableClientState(GL_VERTEX_ARRAY);

    glutSwapBuffers();

    sdkStopTimer(&timer);
    computeFPS();

}

void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo, cuda_vbo_resource);
    }
}


////////////////////////////////////////////////////////////////////////////////
//! Keyboard events handler
////////////////////////////////////////////////////////////////////////////////
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
        case (27) :
            #if defined(__APPLE__) || defined(MACOSX)
                exit(EXIT_SUCCESS);
            #else
                glutDestroyWindow(glutGetWindow());
                return;
            #endif
        case ('r'):
            initial = 0;
            break;
        case ('a'): // left arrow key reduce sextuple strength
            S-=0.1;
            break;
        case ('d'):// right arrow key reduce sextuple strength
            S+=0.1;
            break;
        case ('w'):// up arrow key increase tune
            Qx+=0.01;
            nux = 2.0*3.1415926*Qx;
            break;
        case ('s'):// down arrow key decrease tune
            Qx-=0.01;
            nux = 2.0*3.1415926*Qx;
            break;
        case ('f'):// f key increase second order harmonic
            v2 +=0.01;
            break; 
        case ('v'):// g key decrease second order harmonic
            v2 -=0.01;
            break; 
        case ('g'):// f key increase second order harmonic
            v1 +=0.01;
            break; 
        case ('b'):// g key decrease second order harmonic
            v1 -=0.01;
            break; 
            
    }
}

////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

void motion(int x, int y)
{
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);

    if (mouse_buttons & 1)
    {
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4)
    {
        translate_z += dy * 0.01f;
    }

    mouse_old_x = x;
    mouse_old_y = y;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo)
{
    if (!d_vbo_buffer)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

        // map buffer object
        glBindBuffer(GL_ARRAY_BUFFER, vbo);
        double *data = (double *) glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY);

        // check result
        if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
        {
            // write file for regression test
            sdkWriteFile<double>("./data/regression.dat",
                                data, Npar * 3, 0.0, false);
        }

        // unmap GL buffer object
        if (!glUnmapBuffer(GL_ARRAY_BUFFER))
        {
            fprintf(stderr, "Unmap buffer failed.\n");
            fflush(stderr);
        }

        checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, vbo,
                                                     cudaGraphicsMapFlagsWriteDiscard));

        SDK_CHECK_ERROR_GL();
    }
}
